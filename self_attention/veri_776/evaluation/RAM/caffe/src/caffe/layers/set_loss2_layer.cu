
#include <hip/hip_runtime.h>
// #include <algorithm>
// #include <vector>
// #include "omp.h"
// #include <iostream>
// using namespace std;

// #include "caffe/layers/set_loss2_layer.hpp"
// #include "caffe/util/math_functions.hpp"
// #include "caffe/util/io.hpp"

// namespace caffe
// {
// 	template <typename Dtype>
// 	void SetLoss2Layer<Dtype>::Forward_gpu(
// 		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
// 	{

// 		const Dtype* label = bottom[1]->cpu_data();
// 		const Dtype* data = bottom[0]->cpu_data();
// 		const Dtype* data_gpu = bottom[0]->gpu_data();

// 		// initialization of cat_per_iter_, im_per_cat_ and cat_mean_.

// 		if ( im_per_cat_ < 0 )
// 		{
// 			int j;
// 			for ( j = 0; j < batch_size_; ++j)
// 			{
// 				if (label[0] != label[j])
// 				{
// 					break;
// 				}
// 			}
// 			im_per_cat_ = j;
// 			cat_per_iter_ = batch_size_ / j;
// 			temp_loss_.Reshape(1, 1, cat_per_iter_, code_length_);
// 			cat_mean_.Reshape(1, 1, cat_per_iter_, code_length_);
// 			cat_grad_.Reshape(1, 1, cat_per_iter_, code_length_);
// 		}

// 		// compute cat_mean_
// 		memset(cat_mean_.mutable_cpu_data(), 0, sizeof(Dtype) * code_length_ * cat_per_iter_);
// 		for ( int i = 0; i < batch_size_; i += im_per_cat_)
// 		{
// 			for (int j = 0; j < im_per_cat_; ++j)
// 			{
// 				caffe_gpu_axpby<Dtype>(code_length_, 1. / im_per_cat_, data_gpu + (i + j) * code_length_, 1, cat_mean_.mutable_gpu_data() + int(i / im_per_cat_) * code_length_);
// 			}
// 		}

// 		// compute loss and cat grad
// 		memset(cat_grad_.mutable_cpu_data(), 0, sizeof(Dtype) * code_length_ * cat_per_iter_);
// 		float loss(0);// = new float* [cat_per_iter_];
// 		for ( int i = 0; i < cat_per_iter_; ++i)
// 		{
// 			// loss[i] = new float [cat_per_iter_];
// 			for (int j = i + 1; j < cat_per_iter_; ++j)
// 			{				
// 				caffe_gpu_sub<Dtype>(code_length_, cat_mean_.gpu_data() + i * code_length_, cat_mean_.gpu_data() + j * code_length_, temp_loss_.mutable_gpu_data()+i*code_length_);
// 				Dtype sub_loss;
				
// 				caffe_gpu_dot<Dtype>(code_length_, temp_loss_.gpu_data()+i*code_length_, temp_loss_.gpu_data()+i*code_length_, &sub_loss);
// 				sub_loss = alpha_ - 0.25 * sub_loss;
// 				if (sub_loss > 0)
// 				{
// 					loss += sub_loss;
// 					caffe_gpu_axpby<Dtype>(code_length_, 1., temp_loss_.gpu_data()+i*code_length_, 1., cat_grad_.mutable_gpu_data() + i * code_length_);
// 					caffe_gpu_axpby<Dtype>(code_length_, -1., temp_loss_.gpu_data()+i*code_length_, 1., cat_grad_.mutable_gpu_data() + j * code_length_);
// 				}
// 				// else
// 				// {
// 				// 	loss[i][j] = 0;
// 				// }
// 			}
// 		}

// 		// float loss_sum(0.);
// 		// for (int i = 0; i < cat_per_iter_; ++i)
// 		// {
// 		// 	for (int j = i + 1; j < cat_per_iter_; ++j)
// 		// 	{
// 		// 		loss_sum += loss[i][j];
// 		// 	}
// 		// }
// 		top[0]->mutable_cpu_data()[0] = 2 * loss / cat_per_iter_ / (cat_per_iter_ - 1);
// 	}


// 	template <typename Dtype>
// 	void SetLoss2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
// 		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
// 	{
// 		if (propagate_down[1])
// 		{
// 			LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs. ";
// 		}

// 		Dtype* diff = bottom[0]->mutable_gpu_diff();
// 		loss_weight_ = top[0]->cpu_diff()[0];

// 		caffe_gpu_scal<Dtype>(code_length_ * cat_per_iter_, - loss_weight_ / cat_per_iter_ / (cat_per_iter_ - 1) / im_per_cat_, cat_grad_.mutable_gpu_data());

// 		// copy gradient to each sample
// 		for (int i = 0; i < batch_size_; ++i)
// 		{
// 			cudaMemcpy(diff + i * code_length_, cat_grad_.gpu_data() + int(i / im_per_cat_) * code_length_, sizeof(Dtype) * code_length_, cudaMemcpyDefault);
// 		}
// 	}

//   INSTANTIATE_LAYER_GPU_FUNCS(SetLoss2Layer);

// }