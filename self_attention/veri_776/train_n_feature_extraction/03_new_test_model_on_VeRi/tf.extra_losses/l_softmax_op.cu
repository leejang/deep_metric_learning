
#include <hip/hip_runtime.h>
// MIT License

// Copyright (c) 2018 Changan Wang

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#if GOOGLE_CUDA == 1
#define EIGEN_USE_GPU
#include "l_softmax_op.h"
#include "common.h"

#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_shape.h"

using namespace tensorflow;

#include <cstdint>
#include <cmath>
#include <cfloat>

static __device__ int32_t cuda_factorial(int32_t n){
    int32_t frac = 1;
    while(n-- > 0) frac *= (n+1);
    return frac;
}

template <typename T>
__global__ void LargeMarginSoftmaxCudaKernel(CudaLaunchConfig config, const T * features, const T * weights, const int32_t * global_step, const int32_t * labels,
    const int32_t batch_size, const int32_t num_dimensions, const int32_t output_dimensions,
    const float base, const float gamma, const float power, const float lambda_min, const int32_t margin_order, const bool b_angular,
    float * feat_norm, float * weights_norm, float * cos_theta, float * theta_seg, float * output_lambda, T * losses) {

    *output_lambda = tf_max(base * std::pow(1.f + gamma * global_step[0], -power), lambda_min);//999.1242;//

    for(int32_t index = 0;index < batch_size;++index){
        T temp_sum{0};
        const T *feat_along = features + index * num_dimensions;
        for(int32_t dim_ind = 0;dim_ind < num_dimensions;++dim_ind){
            temp_sum += ldg(feat_along + dim_ind) * ldg(feat_along + dim_ind);
        }
        feat_norm[index] = std::pow(static_cast<float>(temp_sum), .5);
    }
    for(int32_t index = 0;index < output_dimensions;++index){
        T temp_sum{0};
        const T *weights_along = weights + index * num_dimensions;
        for(int32_t dim_ind = 0;dim_ind < num_dimensions;++dim_ind){
            temp_sum += ldg(weights_along + dim_ind) * ldg(weights_along + dim_ind);
        }
        weights_norm[index] = b_angular ? 1. : std::pow(static_cast<float>(temp_sum), .5);
    }
    for(int32_t index = 0;index < margin_order;++index){
        theta_seg[index] = std::cos(_PI * index / margin_order);
    }
    theta_seg[margin_order] = -1.;

    CUDA_1D_KERNEL_LOOP(worker_index, config.virtual_thread_count) {
        const int32_t output_row = worker_index / output_dimensions;
        const int32_t output_col = worker_index % output_dimensions;

        float feat_norm_value = feat_norm[output_row];
        float *p_cos_theta = cos_theta + output_row * output_dimensions;

        const T *feat_start = features + output_row * num_dimensions;
        // get cos_theta for features and all weights rows
        for(int32_t col_ind = 0;col_ind < output_dimensions;++col_ind){
            const T *weights_start = weights + col_ind * num_dimensions;
            T inner_dot{0};
            for(int32_t index = 0;index < num_dimensions;++index){
              inner_dot += ldg(feat_start + index) * ldg(weights_start + index);
            }
            p_cos_theta[col_ind] = static_cast<float>(inner_dot) / (feat_norm[output_row] * weights_norm[col_ind]);
        }
        int32_t k_block = 0;
        for(int32_t index = 1;index < margin_order+1;++index){
          if(p_cos_theta[output_col] > theta_seg[index]){
            k_block = index - 1;
            break;
          }
        }
        float single_cos = p_cos_theta[output_col];
        float cos_n_theta = std::pow(single_cos, margin_order*1.);
        float sin2_theta = 1. - single_cos * single_cos;
        for(int32_t m = 1;m <= margin_order/2; ++m){
          cos_n_theta += std::pow(-1, m) * std::pow(sin2_theta, m * 1.) * std::pow(single_cos, margin_order - 2.*m) * cuda_factorial(margin_order)/(cuda_factorial(2*m)*cuda_factorial(margin_order-2*m)*1.);
        }
        cos_n_theta = std::pow(-1., k_block) * cos_n_theta - 2 * k_block;
        if(ldg(labels + output_row) != output_col) cos_n_theta = 0.;
        *(losses + worker_index) = (feat_norm_value * weights_norm[output_col]) * (p_cos_theta[output_col] * output_lambda[0]/(output_lambda[0] + 1.) + cos_n_theta / (1. + output_lambda[0]));
    }
}

template <typename T>
void LargeMarginSoftmaxFunctor<GPUDevice, T>::operator()(OpKernelContext* context, const GPUDevice& d, typename TTypes<T>::ConstFlat features, typename TTypes<T>::ConstFlat weights, typename TTypes<int32_t>::ConstFlat global_step, typename TTypes<int32_t>::ConstFlat labels,
        const int32_t batch_size, const int32_t num_dimensions, const int32_t output_dimensions,
        const float base, const float gamma, const float power, const float lambda_min, const int32_t margin_order, const bool b_angular,
        typename TTypes<float>::Flat feat_norm, typename TTypes<float>::Flat weights_norm,
        typename TTypes<float>::Flat cos_theta, typename TTypes<float>::Flat theta_seg,
        typename TTypes<float>::Flat output_lambda, typename TTypes<T>::Flat losses) {

    CudaLaunchConfig config = GetCudaLaunchConfig(batch_size * output_dimensions, d);
    LargeMarginSoftmaxCudaKernel <<<config.block_count,
                        config.thread_per_block, 0, d.stream()>>> (config, features.data(), weights.data(), global_step.data(), labels.data(),
                            batch_size, num_dimensions, output_dimensions, base, gamma, power, lambda_min, margin_order, b_angular,
                            feat_norm.data(), weights_norm.data(), cos_theta.data(), theta_seg.data(), output_lambda.data(), losses.data());

    cudaError_t err = cudaGetLastError();
    if(cudaSuccess != err)
    {
      fprintf( stderr, "cudaCheckError() failed : %s\n", cudaGetErrorString( err ) );
      exit( -1 );
    }
}

template struct LargeMarginSoftmaxFunctor<GPUDevice, float>;
// #define DEFINE_GPU_SPECS(T)   \
//   template struct LargeMarginSoftmaxFunctorGPU<T>;

// TF_CALL_GPU_NUMBER_TYPES(DEFINE_GPU_SPECS);

#endif  // GOOGLE_CUDA
