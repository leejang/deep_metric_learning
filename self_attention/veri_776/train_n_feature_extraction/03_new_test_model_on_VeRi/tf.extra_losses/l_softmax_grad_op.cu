
#include <hip/hip_runtime.h>
// MIT License

// Copyright (c) 2018 Changan Wang

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#if GOOGLE_CUDA == 1
#define EIGEN_USE_GPU
#include "l_softmax_op.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_shape.h"

using namespace tensorflow;

#include <cstdint>
#include <cmath>
#include <cfloat>

static __device__ int32_t cuda_factorial(int32_t n){
    int32_t frac = 1;
    while(n-- > 0) frac *= (n+1);
    return frac;
}

// Define the CUDA kernel.
template <typename T>
__global__ void LargeMarginSoftmaxGradCudaKernel(CudaLaunchConfig config, const T * back_grads, const T * features, const T * weights, const float * cur_lambda, const int32_t * labels,
    const int32_t batch_size, const int32_t num_dimensions, const int32_t output_dimensions, const int32_t margin_order, const bool b_angular,
    float * feat_norm, float * weights_norm, float * cos_theta, float * theta_seg, T * grad_features, T * grad_weights) {

    for(int32_t index = 0;index < batch_size;++index){
        T temp_sum{0};
        const T *feat_along = features + index * num_dimensions;
        for(int32_t dim_ind = 0;dim_ind < num_dimensions;++dim_ind){
            temp_sum += ldg(feat_along + dim_ind) * ldg(feat_along + dim_ind);
        }
        feat_norm[index] = std::pow(static_cast<float>(temp_sum), .5);
    }
    for(int32_t index = 0;index < output_dimensions;++index){
        T temp_sum{0};
        const T *weights_along = weights + index * num_dimensions;
        for(int32_t dim_ind = 0;dim_ind < num_dimensions;++dim_ind){
            temp_sum += ldg(weights_along + dim_ind) * ldg(weights_along + dim_ind);
        }
        weights_norm[index] = b_angular ? 1. : std::pow(static_cast<float>(temp_sum), .5);
    }
    for(int32_t index = 0;index < margin_order;++index){
        theta_seg[index] = std::cos(_PI * index / margin_order);
    }
    theta_seg[margin_order] = -1.;

    CUDA_1D_KERNEL_LOOP(worker_index, config.virtual_thread_count) {
        const int32_t output_row = worker_index / output_dimensions;
        const int32_t output_col = worker_index % output_dimensions;

        float feat_norm_value = feat_norm[output_row];
        float *p_cos_theta = cos_theta + output_row * output_dimensions;

        const T *feat_start = features + output_row * num_dimensions;
        // get cos_theta for features and all weights rows
        for(int32_t col_ind = 0;col_ind < output_dimensions;++col_ind){
            const T *weights_start = weights + col_ind * num_dimensions;
            T inner_dot{0};
            for(int32_t index = 0;index < num_dimensions;++index){
              inner_dot += ldg(feat_start + index) * ldg(weights_start + index);
            }
            p_cos_theta[col_ind] = static_cast<float>(inner_dot) / (feat_norm[output_row] * weights_norm[col_ind]);
        }
        int32_t k_block = 0;
        for(int32_t index = 1;index < margin_order+1;++index){
          if(p_cos_theta[output_col] > theta_seg[index]){
            k_block = index - 1;
            break;
          }
        }

        float single_cos = p_cos_theta[output_col];
        float sin2_theta = 1. - single_cos * single_cos;
        float cos_n_theta = 0.;
        // calculate cons_n_theta
        if(ldg(labels+output_row) == output_col){
          cos_n_theta = std::pow(single_cos, margin_order*1.);
          for(int32_t m = 1; m <= margin_order / 2; ++m){
            float binomial = cuda_factorial(margin_order) / (cuda_factorial(2 * m) * cuda_factorial(margin_order - 2 * m) * 1.);
            cos_n_theta += std::pow(-1, m) * std::pow(sin2_theta, m * 1.) * std::pow(single_cos, margin_order - 2. * m) * binomial;
          }
          cos_n_theta = std::pow(-1., k_block) * cos_n_theta - 2 * k_block;
        }
        // grad of cos_n_theta by cos_theta
        float grad_of_cos_theta = margin_order * std::pow(single_cos, margin_order - 1.);
        for(int32_t m = 1; m <= margin_order / 2; ++m){
          float binomial = cuda_factorial(margin_order) / (cuda_factorial(2 * m) * cuda_factorial(margin_order - 2 * m) * 1.);
          grad_of_cos_theta += std::pow(-1, m) * std::pow(sin2_theta, m - 1.) * std::pow(single_cos, margin_order - 2 * m - 1.) * (-2 * m + margin_order - margin_order * std::pow(single_cos, 2.)) * binomial;
        }
        grad_of_cos_theta = grad_of_cos_theta * std::pow(-1., k_block);
        // backprop
        const float input_grad = ldg(back_grads + worker_index);
        const T *weights_start = weights + output_col * num_dimensions;

        T *grad_feat_start = grad_features + output_row * num_dimensions;
        T *grad_weights_start = grad_weights + output_col * num_dimensions;
        // softmax
        for(int32_t dim_ind = 0; dim_ind < num_dimensions; ++dim_ind){
          atomicAdd(grad_weights_start + dim_ind, input_grad * ldg(cur_lambda)/(ldg(cur_lambda) + 1.) * ldg(feat_start+dim_ind));
          atomicAdd(grad_feat_start + dim_ind, input_grad * ldg(cur_lambda)/(ldg(cur_lambda) + 1.) * ldg(weights_start+dim_ind));
        }
        // large margin softmax
        if(ldg(labels + output_row) == output_col){
          for(int32_t dim_ind = 0; dim_ind < num_dimensions; ++dim_ind){
            float wx_norm = feat_norm_value * weights_norm[output_col];

            float grad_cos_n_theta_by_w = b_angular ? grad_of_cos_theta * feat_start[dim_ind] / feat_norm_value : grad_of_cos_theta / (feat_norm_value * weights_norm[output_col] * weights_norm[output_col]) *
                                          ( (ldg(feat_start+dim_ind) * weights_norm[output_col]) -
                                            (wx_norm * single_cos * ldg(weights_start+dim_ind) / weights_norm[output_col])
                                          );
            if(b_angular){
                atomicAdd(grad_weights_start + dim_ind, input_grad * feat_norm_value/(ldg(cur_lambda) + 1.) *grad_cos_n_theta_by_w );
            }else{
                atomicAdd(grad_weights_start + dim_ind, input_grad * feat_norm_value/(ldg(cur_lambda) + 1.) * (
                                          cos_n_theta * ldg(weights_start+dim_ind) / weights_norm[output_col] +
                                          grad_cos_n_theta_by_w * weights_norm[output_col]       ) );
            }

            float grad_cos_n_theta_by_x = grad_of_cos_theta / (weights_norm[output_col] * feat_norm_value * feat_norm_value) *
                                          ( (ldg(weights_start+dim_ind) * feat_norm_value) -
                                            (wx_norm * single_cos * ldg(feat_start+dim_ind) / feat_norm_value)
                                          );

            atomicAdd(grad_feat_start + dim_ind, input_grad * weights_norm[output_col]/(ldg(cur_lambda) + 1.) * (
                                          cos_n_theta * ldg(feat_start+dim_ind) / feat_norm_value +
                                          grad_cos_n_theta_by_x * feat_norm_value       ) );
          }
        }
    }
}


template <typename T>
void LargeMarginSoftmaxGradFunctor<GPUDevice, T>::operator()(OpKernelContext* context, const GPUDevice& d, typename TTypes<T>::ConstFlat back_grads, typename TTypes<T>::ConstFlat features, typename TTypes<T>::ConstFlat weights, typename TTypes<float>::ConstFlat cur_lambda, typename TTypes<int32_t>::ConstFlat labels,
        const int32_t batch_size, const int32_t num_dimensions, const int32_t output_dimensions, const int32_t margin_order, const bool b_angular,
        typename TTypes<float>::Flat feat_norm, typename TTypes<float>::Flat weights_norm,
        typename TTypes<float>::Flat cos_theta, typename TTypes<float>::Flat theta_seg,
        typename TTypes<T>::Flat grad_features, typename TTypes<T>::Flat grad_weights) {

    CudaLaunchConfig config = GetCudaLaunchConfig(batch_size * num_dimensions, d);
    SetZero <<<config.block_count, config.thread_per_block, 0, d.stream()>>> (batch_size * num_dimensions, grad_features.data());
    config = GetCudaLaunchConfig(output_dimensions * num_dimensions, d);
    SetZero <<<config.block_count, config.thread_per_block, 0, d.stream()>>> (output_dimensions * num_dimensions, grad_weights.data());

    config = GetCudaLaunchConfig(batch_size * output_dimensions, d);
    LargeMarginSoftmaxGradCudaKernel <<<config.block_count,
                        config.thread_per_block, 0, d.stream()>>> (config, back_grads.data(), features.data(), weights.data(), cur_lambda.data(), labels.data(),
                            batch_size, num_dimensions, output_dimensions, margin_order, b_angular,
                            feat_norm.data(), weights_norm.data(), cos_theta.data(), theta_seg.data(), grad_features.data(), grad_weights.data());

    cudaError_t err = cudaGetLastError();
    if(cudaSuccess != err)
    {
      fprintf( stderr, "cudaCheckError() failed : %s\n", cudaGetErrorString( err ) );
      exit( -1 );
    }
}

template struct LargeMarginSoftmaxGradFunctor<GPUDevice, float>;
// #define DEFINE_GPU_SPECS(T)   \
//   template struct LargeMarginSoftmaxGradFunctor<T>;

// TF_CALL_GPU_NUMBER_TYPES(DEFINE_GPU_SPECS);

#endif  // GOOGLE_CUDA
